#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

#include "hipsparse.h"
#include "hipsolver.h"

/* Cuda 7.5 preview */
#include "hipsolver.h"

#include "hip/hip_runtime_api.h"
#include "helper_cusolver.h"

#define integer_t int
#define complex_t float
#define real_t float

#define SOLVE_ON_GPU 1
#define CUSOLVER_NO_REORDERING 0

int main(int argc, char const *argv[])
{
	/* definition of a small matrix */
	const integer_t n     = 4;
	const integer_t nnz   = 9;
	const integer_t nrhs  = 1;

	/* additional variables related with the solver */
	integer_t issym    = 0;
	real_t    epsilon  = 1e-5;
	integer_t singular = 0;
	integer_t reorder  = CUSOLVER_NO_REORDERING;

	/* cuSolver handler */
	hipsolverSpHandle_t cusolverHandle = NULL;

	/* cusparse handler */
	hipsparseHandle_t cusparseHandle = NULL;

	/* cuSparse matrix descriptor */
	hipsparseMatDescr_t MatDescr = NULL;

	complex_t *d_aij;
	integer_t *d_colind;
	integer_t *d_rowptr;
	complex_t *d_xij;
	complex_t *d_bij;

	/* create a dummy csr matrix */
	complex_t aij   [9] = {2., 4., 6., 1., 2., 4., 5., 1., 3. };
	integer_t colind[9] = {0, 1, 2, 1, 2, 1, 2, 3, 3 };
	integer_t rowptr[5] = {0, 3, 5, 8, 9};

	/* define solution and rhs vectors */
	complex_t xij[4] = {0.};
	complex_t bij[4] = {1.};

	/* allocate space on the device memory */
	checkCudaErrors( hipMalloc((void**) &d_aij   ,  nnz  * sizeof(complex_t)) );
	checkCudaErrors( hipMalloc((void**) &d_colind,  nnz  * sizeof(integer_t)) );
	checkCudaErrors( hipMalloc((void**) &d_rowptr, (n+1) * sizeof(integer_t)) );
	checkCudaErrors( hipMalloc((void**) &d_xij, n * nrhs * sizeof(complex_t)) );
	checkCudaErrors( hipMalloc((void**) &d_bij, n * nrhs * sizeof(complex_t)) );

	/* transfer arrays to the device memory */
	checkCudaErrors( hipMemcpy( d_aij   , aij   ,  nnz     * sizeof(complex_t), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy( d_colind, colind,  nnz     * sizeof(integer_t), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy( d_rowptr, rowptr, (n+1)    * sizeof(integer_t), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy( d_xij   , xij   , n * nrhs * sizeof(complex_t), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy( d_bij   , bij   , n * nrhs * sizeof(complex_t), hipMemcpyHostToDevice ) );


	/* The handle must be initialized prior to calling any other library function */
	checkCudaErrors( hipsolverSpCreate(&cusolverHandle));

	/* Create matrix descriptor */
	checkCudaErrors( hipsparseSetMatType      ( MatDescr, HIPSPARSE_MATRIX_TYPE_GENERAL ) );
	checkCudaErrors( hipsparseSetMatIndexBase ( MatDescr, HIPSPARSE_INDEX_BASE_ZERO     ) );

	/* Verify if A is symmetric or not */
	checkCudaErrors( cusolverSpXcsrissymHost( cusolverHandle, 
		n, 
		nnz, 
		MatDescr, 
		rowptr, 
		rowptr +1, 
		colind, 
		&issym ));

	if ( issym ) fprintf(stderr, "Input matrix is symmetric.\n");

	/* Solve the linear system on the GPU using the high-level API    */
	/* We use QR factorization, LU factorization is not supported yet */
	checkCudaErrors( hipsolverSpScsrlsvqr ( cusolverHandle,
	                n,
	                nnz,
	                MatDescr,
	                d_aij,
	                d_rowptr,
	                d_colind,
	                d_bij,
	                epsilon,
	                reorder,
	                d_xij,
	                &singular));

	/* transfer the solution back to the host memory */
	checkCudaErrors( hipMemcpy( xij, d_xij, n * nrhs * sizeof(complex_t), hipMemcpyDeviceToHost ));

	fprintf(stderr, "\nSolution of the linear system\n");
	for(integer_t i=0; i < n; i++ )
		fprintf(stderr, "x[%d] = %.6f\n", i, xij[i]);


	/* NOW, WE USE THE LOW-LEVEL API */
	csrqrInfo_t csrqrInfo = NULL;
	checkCudaErrors( cusolverSpCreateCsrqrInfo ( &csrqrInfo ));

	/* analyses sparsity pattern of H and Q matrices */
	checkCudaErrors( cusolverSpXcsrqrAnalysis ( cusolverHandle,
                           n,
                           n,
                           nnz,
                           MatDescr,
                           d_rowptr,
                           d_colind,
                           csrqrInfo ));

	/* After the analysis, the size of working space to perform QR factorization can be retrieved */
	size_t internalDataInBytes = 0;
	size_t workspaceInBytes    = 0;

	checkCudaErrors( cusolverSpScsrqrBufferInfo ( cusolverHandle,
                           n,
                           n,
                           nnz,
                           MatDescr,
                           d_aij,
                           d_rowptr,
                           d_colind,
                           csrqrInfo,
                           &internalDataInBytes,
                           &workspaceInBytes));

	fprintf(stderr, "\nInternal data in bytes : %lu", internalDataInBytes);
	fprintf(stderr, "\nWorkspace in bytes     : %lu", workspaceInBytes   );

	/* allocate space for the later factorization */
	void *d_work;
	checkCudaErrors( hipMalloc((void**) &d_work, workspaceInBytes ));

	/* This function shifts diagonal of A by parameter mu such that we can factorize */
	/* For linear solver, the user just sets mu to zero.                             */ 
	/* For eigenvalue solver, mu can be a value of shift in inverse-power method.    */
	checkCudaErrors( cusolverSpScsrqrSetup ( cusolverHandle,
                           n,
                           n,
                           nnz,
                           MatDescr,
                           d_aij,
                           d_rowptr,
                           d_colind,
                           0,
                           csrqrInfo ));

	/* Perform numerical factorization */
	/* If either x or b is nil, only factorization is done. The user needs cusolverSpXcsrqrSolve    */
	/* to find the least-square solution.                                                           */
	/* If both x and b are not nil, QR factorization and solve are combined together. b is over-    */
	/* written by c and x is the solution of least-square.                                          */
	/* pBuffer: buffer allocated by the user, the size is returned by cusolverSpXcsrqrBufferInfo(). */
	checkCudaErrors( cusolverSpScsrqrFactor ( cusolverHandle,
                           n,
                           n,
                           nnz,
                           d_bij,
                           d_xij,
                           csrqrInfo,
                           d_work ));

//	/* Solve the system, only if it was only factorized */
//	checkCudaErrors( cusolverSpScsrqrSolve ( cusolverHandle,
//                           m,
//                           n,
//                           d_bij,
//                           d_xij,
//                           csrqrInfo,
//                           d_work ));

	/* transfer the solution back to the host memory */
	checkCudaErrors( hipMemcpy( xij, d_xij, n * nrhs * sizeof(complex_t), hipMemcpyDeviceToHost ));

	fprintf(stderr, "\nSolution of the linear system\n");
	for(integer_t i=0; i < n; i++ )
		fprintf(stderr, "x[%d] = %.6f\n", i, xij[i]);


	checkCudaErrors( cusolverSpDestroyCsrqrInfo ( csrqrInfo ));
	if ( d_work ) { hipFree( d_work ); }





	/* clean up and */
	if ( cusolverHandle ) { checkCudaErrors( hipsolverSpDestroy(cusolverHandle)); }
	if ( cusparseHandle ) { checkCudaErrors( hipsparseDestroy  (cusparseHandle)); }
	if ( MatDescr       ) { checkCudaErrors( hipsparseDestroyMatDescr(MatDescr)); }

	if ( d_aij    ) { checkCudaErrors( hipFree( d_aij    )); }
	if ( d_colind ) { checkCudaErrors( hipFree( d_colind )); }
	if ( d_rowptr ) { checkCudaErrors( hipFree( d_rowptr )); }
	if ( d_xij    ) { checkCudaErrors( hipFree( d_xij    )); }
	if ( d_bij    ) { checkCudaErrors( hipFree( d_bij    )); }

	hipDeviceReset();


	fprintf(stderr, "Testing cuSolver High Level Interface\n");

	return 0;
}